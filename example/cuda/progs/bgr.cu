
#include <hip/hip_runtime.h>
#define NUMTHREADS 384

__global__ void FragmentProgram(int size, unsigned char *in, unsigned char *out) {
  int i=0;
  size=size/3;

  for(i=threadIdx.x; i<size; i=i+NUMTHREADS) {
    out[i*3]=in[i*3+2];
    out[i*3+1]=in[i*3+1];
    out[i*3+2]=in[i*3];
  }
}


