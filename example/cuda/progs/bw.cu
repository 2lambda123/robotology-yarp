
#include <hip/hip_runtime.h>
#define NUMTHREADS 384

__global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
  int i=0;

  for(i=threadIdx.x; i<w*h; i=i+NUMTHREADS) {
    out[i*3]=(in[i*3]+in[i*3+1]+in[i*3+2])/3;
    out[i*3+1]=out[i*3];
    out[i*3+2]=out[i*3];
  }
}


