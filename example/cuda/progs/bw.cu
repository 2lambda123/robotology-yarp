#include "hip/hip_runtime.h"
#define NUMTHREADS 32
#define NUMBLOCKS 96

__global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
  int i=0;

  for(i=threadIdx.x+blockIdx.x*blockDim.x; i<parms.n; i+=blockDim.x*gridDim.x) {
    out[i*3]=(in[i*3]+in[i*3+1]+in[i*3+2])/3;
    out[i*3+1]=out[i*3];
    out[i*3+2]=out[i*3];
  }
}


