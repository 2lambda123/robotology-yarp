#include "hip/hip_runtime.h"
#define NUMTHREADS 384

__global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
  int i=0;

  for(i=threadIdx.x+blockIdx.x*ATHREAD_COUNT; i<w*h; i+=ATHREAD_COUNT*BLOCKS) {
    out[i*3]=(in[i*3]+in[i*3+1]+in[i*3+2])/3;
    out[i*3+1]=out[i*3];
    out[i*3+2]=out[i*3];
  }
}


