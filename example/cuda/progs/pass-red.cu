
#include <hip/hip_runtime.h>
#define ATHREAD_COUNT 32
#define BLOCKS 96

__global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
  int i=0;

  for(i=threadIdx.x+blockIdx.x*ATHREAD_COUNT; i<w*h; i+=ATHREAD_COUNT*BLOCKS) {
    if(in[i*3]>=200) {
      //out[i*3]=255;
      //out[i*3+1]=255;
      //out[i*3+2]=255;
    } else {
      out[i*3]=0;
      out[i*3+1]=0;
      out[i*3+2]=0;
    }
  }
}


