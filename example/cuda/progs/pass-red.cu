
#include <hip/hip_runtime.h>
#define NUMTHREADS 384

__global__ void FragmentProgram(int size, unsigned char *in, unsigned char *out) {
  int i=0;
  size=size/3;

  for(i=threadIdx.x; i<size; i=i+NUMTHREADS) {
    if(in[i*3]>=200) {
      //out[i*3]=255;
      //out[i*3+1]=255;
      //out[i*3+2]=255;
    } else {
      out[i*3]=0;
      out[i*3+1]=0;
      out[i*3+2]=0;
    }
  }
}


