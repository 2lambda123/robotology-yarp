
#include <hip/hip_runtime.h>
#define NUMTHREADS 384

__global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
  int i=0;

  for(i=threadIdx.x; i<w*h; i=i+NUMTHREADS) {
    if(in[i*3]>=200) {
      //out[i*3]=255;
      //out[i*3+1]=255;
      //out[i*3+2]=255;
    } else {
      out[i*3]=0;
      out[i*3+1]=0;
      out[i*3+2]=0;
    }
  }
}


